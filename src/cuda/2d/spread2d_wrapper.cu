#include "hip/hip_runtime.h"
#include <cassert>
#include <iomanip>
#include <iostream>

#include <hip/hip_complex.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include <cufinufft/memtransfer.h>
#include <cufinufft/precision_independent.h>
#include <cufinufft/spreadinterp.h>
using namespace cufinufft::common;
using namespace cufinufft::memtransfer;

namespace cufinufft {
namespace spreadinterp {

int CUFINUFFT_SPREAD2D(int nf1, int nf2, CUCPX *d_fw, int M, CUFINUFFT_FLT *d_kx, CUFINUFFT_FLT *d_ky, CUCPX *d_c,
                       CUFINUFFT_PLAN d_plan)
/*
    This c function is written for only doing 2D spreading. See
    test/spread2d_test.cu for usage.

    Melody Shih 07/25/19
    not allocate,transfer and free memories on gpu. Shih 09/24/20
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    d_plan->kx = d_kx;
    d_plan->ky = d_ky;
    d_plan->c = d_c;
    d_plan->fw = d_fw;

    int ier;
    d_plan->nf1 = nf1;
    d_plan->nf2 = nf2;
    d_plan->M = M;
    d_plan->maxbatchsize = 1;

    hipEventRecord(start);
    ier = ALLOCGPUMEM2D_PLAN(d_plan);
    ier = ALLOCGPUMEM2D_NUPTS(d_plan);

    if (d_plan->opts.gpu_method == 1) {
        ier = CUSPREAD2D_NUPTSDRIVEN_PROP(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_nuptsdriven_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

    if (d_plan->opts.gpu_method == 2) {
        ier = CUSPREAD2D_SUBPROB_PROP(nf1, nf2, M, d_plan);
        if (ier != 0) {
            printf("error: cuspread2d_subprob_prop, method(%d)\n", d_plan->opts.gpu_method);
            return ier;
        }
    }

#ifdef TIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Obtain Spread Prop\t %.3g ms\n", milliseconds);
#endif

    hipEventRecord(start);
    ier = CUSPREAD2D(d_plan, 1);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Spread (%d)\t\t %5.3f ms\n", d_plan->opts.gpu_method, milliseconds);
#endif

    hipEventRecord(start);
    FREEGPUMEMORY2D(d_plan);
#ifdef TIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] Free GPU memory\t %.3g ms\n", milliseconds);
#endif
    return ier;
}

int CUSPREAD2D(CUFINUFFT_PLAN d_plan, int blksize)
/*
    A wrapper for different spreading methods.

    Methods available:
    (1) Non-uniform points driven
    (2) Subproblem

    Melody Shih 07/25/19
*/
{
    int nf1 = d_plan->nf1;
    int nf2 = d_plan->nf2;
    int M = d_plan->M;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ier;
    switch (d_plan->opts.gpu_method) {
    case 1: {
        hipEventRecord(start);
        ier = CUSPREAD2D_NUPTSDRIVEN(nf1, nf2, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread2d_gpu_nuptsdriven" << std::endl;
            return 1;
        }
    } break;
    case 2: {
        hipEventRecord(start);
        ier = CUSPREAD2D_SUBPROB(nf1, nf2, M, d_plan, blksize);
        if (ier != 0) {
            std::cout << "error: cnufftspread2d_gpu_subprob" << std::endl;
            return 1;
        }
    } break;
    default:
        std::cout << "error: incorrect method, should be 1,2,3" << std::endl;
        return 2;
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "[time  ]"
              << " Spread " << milliseconds << " ms" << std::endl;
#endif
    return ier;
}

int CUSPREAD2D_NUPTSDRIVEN_PROP(int nf1, int nf2, int M, CUFINUFFT_PLAN d_plan) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    if (d_plan->opts.gpu_sort) {

        int bin_size_x = d_plan->opts.gpu_binsizex;
        int bin_size_y = d_plan->opts.gpu_binsizey;
        if (bin_size_x < 0 || bin_size_y < 0) {
            std::cout << "error: invalid binsize (binsizex, binsizey) = (";
            std::cout << bin_size_x << "," << bin_size_y << ")" << std::endl;
            return 1;
        }

        int numbins[2];
        numbins[0] = ceil((CUFINUFFT_FLT)nf1 / bin_size_x);
        numbins[1] = ceil((CUFINUFFT_FLT)nf2 / bin_size_y);

#ifdef DEBUG
        std::cout << "[debug ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
                  << d_plan->opts.gpu_binsizey << "]" << std::endl;
        std::cout << "[debug ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

        CUFINUFFT_FLT *d_kx = d_plan->kx;
        CUFINUFFT_FLT *d_ky = d_plan->ky;
#ifdef DEBUG
        CUFINUFFT_FLT *h_kx;
        CUFINUFFT_FLT *h_ky;
        h_kx = (CUFINUFFT_FLT *)malloc(M * sizeof(CUFINUFFT_FLT));
        h_ky = (CUFINUFFT_FLT *)malloc(M * sizeof(CUFINUFFT_FLT));

        checkCudaErrors(hipMemcpy(h_kx, d_kx, M * sizeof(CUFINUFFT_FLT), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(h_ky, d_ky, M * sizeof(CUFINUFFT_FLT), hipMemcpyDeviceToHost));
        for (int i = M - 10; i < M; i++) {
            std::cout << "[debug ] ";
            std::cout << "(" << std::setw(3) << h_kx[i] << "," << std::setw(3) << h_ky[i] << ")" << std::endl;
        }
#endif
        int *d_binsize = d_plan->binsize;
        int *d_binstartpts = d_plan->binstartpts;
        int *d_sortidx = d_plan->sortidx;
        int *d_idxnupts = d_plan->idxnupts;

        int pirange = d_plan->spopts.pirange;

        hipEventRecord(start);
        checkCudaErrors(hipMemset(d_binsize, 0, numbins[0] * numbins[1] * sizeof(int)));
        CalcBinSize_noghost_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0],
                                                                numbins[1], d_binsize, d_kx, d_ky, d_sortidx, pirange);
#ifdef SPREADTIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_binsize; // For debug
        h_binsize = (int *)malloc(numbins[0] * numbins[1] * sizeof(int));
        checkCudaErrors(
            hipMemcpy(h_binsize, d_binsize, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
        std::cout << "[debug ] bin size:" << std::endl;
        for (int j = 0; j < numbins[1]; j++) {
            std::cout << "[debug ] ";
            for (int i = 0; i < numbins[0]; i++) {
                if (i != 0)
                    std::cout << " ";
                std::cout << " bin[" << std::setw(1) << i << "," << std::setw(1) << j << "]=" << h_binsize[i + j * numbins[0]];
            }
            std::cout << std::endl;
        }
        free(h_binsize);
        std::cout << "[debug ] ------------------------------------------------" << std::endl;

        int *h_sortidx;
        h_sortidx = (int *)malloc(M * sizeof(int));

        checkCudaErrors(hipMemcpy(h_sortidx, d_sortidx, M * sizeof(int), hipMemcpyDeviceToHost));

        for (int i = 0; i < M; i++) {
            if (h_sortidx[i] < 0) {
                std::cout << "[debug ] ";
                std::cout << "point[" << std::setw(3) << i << "]=" << std::setw(3) << h_sortidx[i] << std::endl;
                std::cout << "[debug ] ";
                printf("(%10.10f, %10.10f) ", RESCALE(h_kx[i], nf1, pirange), RESCALE(h_ky[i], nf1, pirange));
                printf("(%10.10f, %10.10f) ", RESCALE(h_kx[i], nf1, pirange) / 32, RESCALE(h_ky[i], nf1, pirange) / 32);
                printf("(%f, %f)\n", floor(RESCALE(h_kx[i], nf1, pirange) / 32),
                       floor(RESCALE(h_ky[i], nf1, pirange) / 32));
            }
        }
#endif
        hipEventRecord(start);
        int n = numbins[0] * numbins[1];
        thrust::device_ptr<int> d_ptr(d_binsize);
        thrust::device_ptr<int> d_result(d_binstartpts);
        thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_binstartpts;
        h_binstartpts = (int *)malloc((numbins[0] * numbins[1]) * sizeof(int));
        checkCudaErrors(
            hipMemcpy(h_binstartpts, d_binstartpts, (numbins[0] * numbins[1]) * sizeof(int), hipMemcpyDeviceToHost));
        std::cout << "[debug ] Result of scan bin_size array:" << std::endl;
        for (int j = 0; j < numbins[1]; j++) {
            std::cout << "[debug ] ";
            for (int i = 0; i < numbins[0]; i++) {
                if (i != 0)
                    std::cout << " ";
                std::cout << " bin[" << std::setw(1) << i << "," << std::setw(1) << j
                          << "]=" << h_binstartpts[i + j * numbins[0]];
            }
            std::cout << std::endl;
        }
        free(h_binstartpts);
        std::cout << "[debug ] ------------------------------------------------" << std::endl;
#endif
        hipEventRecord(start);
        CalcInvertofGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, bin_size_x, bin_size_y, numbins[0], numbins[1],
                                                                      d_binstartpts, d_sortidx, d_kx, d_ky, d_idxnupts,
                                                                      pirange, nf1, nf2);
#ifdef SPREADTIME
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel CalcInvertofGlobalSortIdx_2d \t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
        int *h_idxnupts;
        h_idxnupts = (int *)malloc(M * sizeof(int));
        checkCudaErrors(hipMemcpy(h_idxnupts, d_idxnupts, M * sizeof(int), hipMemcpyDeviceToHost));
        for (int i = 0; i < 10; i++) {
            std::cout << "[debug ] idx=" << h_idxnupts[i] << std::endl;
        }
        free(h_idxnupts);
#endif
    } else {
        int *d_idxnupts = d_plan->idxnupts;

        hipEventRecord(start);
        TrivialGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, d_idxnupts);
#ifdef SPREADTIME
        float milliseconds = 0;
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("[time  ] \tKernel TrivialGlobalSortIDx_2d \t\t%.3g ms\n", milliseconds);
#endif
    }
    return 0;
}

int CUSPREAD2D_NUPTSDRIVEN(int nf1, int nf2, int M, CUFINUFFT_PLAN d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 threadsPerBlock;
    dim3 blocks;

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    int pirange = d_plan->spopts.pirange;
    int *d_idxnupts = d_plan->idxnupts;
    CUFINUFFT_FLT es_c = d_plan->spopts.ES_c;
    CUFINUFFT_FLT es_beta = d_plan->spopts.ES_beta;
    CUFINUFFT_FLT sigma = d_plan->spopts.upsampfac;

    CUFINUFFT_FLT *d_kx = d_plan->kx;
    CUFINUFFT_FLT *d_ky = d_plan->ky;
    CUCPX *d_c = d_plan->c;
    CUCPX *d_fw = d_plan->fw;

    threadsPerBlock.x = 16;
    threadsPerBlock.y = 1;
    blocks.x = (M + threadsPerBlock.x - 1) / threadsPerBlock.x;
    blocks.y = 1;
    hipEventRecord(start);
    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_NUptsdriven_Horner<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M,
                                                                      ns, nf1, nf2, sigma, d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_NUptsdriven<<<blocks, threadsPerBlock>>>(d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns,
                                                               nf1, nf2, es_c, es_beta, d_idxnupts, pirange);
        }
    }

#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Spread_2d_NUptsdriven (%d)\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}
int CUSPREAD2D_SUBPROB_PROP(int nf1, int nf2, int M, CUFINUFFT_PLAN d_plan)
/*
    This function determines the properties for spreading that are independent
    of the strength of the nodes,  only relates to the locations of the nodes,
    which only needs to be done once.
*/
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    if (bin_size_x < 0 || bin_size_y < 0) {
        std::cout << "error: invalid binsize (binsizex, binsizey) = (";
        std::cout << bin_size_x << "," << bin_size_y << ")" << std::endl;
        return 1;
    }
    int numbins[2];
    numbins[0] = ceil((CUFINUFFT_FLT)nf1 / bin_size_x);
    numbins[1] = ceil((CUFINUFFT_FLT)nf2 / bin_size_y);
#ifdef DEBUG
    std::cout << "[debug  ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
              << d_plan->opts.gpu_binsizey << "]" << std::endl;
    std::cout << "[debug  ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

    CUFINUFFT_FLT *d_kx = d_plan->kx;
    CUFINUFFT_FLT *d_ky = d_plan->ky;

#ifdef DEBUG
    CUFINUFFT_FLT *h_kx;
    CUFINUFFT_FLT *h_ky;
    h_kx = (CUFINUFFT_FLT *)malloc(M * sizeof(CUFINUFFT_FLT));
    h_ky = (CUFINUFFT_FLT *)malloc(M * sizeof(CUFINUFFT_FLT));

    checkCudaErrors(hipMemcpy(h_kx, d_kx, M * sizeof(CUFINUFFT_FLT), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_ky, d_ky, M * sizeof(CUFINUFFT_FLT), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "[debug ]";
        std::cout << "(" << std::setw(3) << h_kx[i] << "," << std::setw(3) << h_ky[i] << ")" << std::endl;
    }
#endif
    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_sortidx = d_plan->sortidx;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int *d_subprob_to_bin = NULL;

    int pirange = d_plan->spopts.pirange;

    hipEventRecord(start);
    checkCudaErrors(hipMemset(d_binsize, 0, numbins[0] * numbins[1] * sizeof(int)));
    CalcBinSize_noghost_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, nf1, nf2, bin_size_x, bin_size_y, numbins[0], numbins[1],
                                                            d_binsize, d_kx, d_ky, d_sortidx, pirange);
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel CalcBinSize_noghost_2d \t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_binsize; // For debug
    h_binsize = (int *)malloc(numbins[0] * numbins[1] * sizeof(int));
    checkCudaErrors(hipMemcpy(h_binsize, d_binsize, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] bin size:" << std::endl;
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << " bin[" << std::setw(3) << i << "," << std::setw(3) << j << "]=" << h_binsize[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_binsize);
    std::cout << "[debug ] ----------------------------------------------------" << std::endl;
#endif
#ifdef DEBUG
    int *h_sortidx;
    h_sortidx = (int *)malloc(M * sizeof(int));
    checkCudaErrors(hipMemcpy(h_sortidx, d_sortidx, M * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ]";
    for (int i = 0; i < M; i++) {
        std::cout << "[debug] point[" << std::setw(3) << i << "]=" << std::setw(3) << h_sortidx[i] << std::endl;
    }

#endif

    hipEventRecord(start);
    int n = numbins[0] * numbins[1];
    thrust::device_ptr<int> d_ptr(d_binsize);
    thrust::device_ptr<int> d_result(d_binstartpts);
    thrust::exclusive_scan(d_ptr, d_ptr + n, d_result);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel BinStartPts_2d \t\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_binstartpts;
    h_binstartpts = (int *)malloc((numbins[0] * numbins[1]) * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_binstartpts, d_binstartpts, (numbins[0] * numbins[1]) * sizeof(int), hipMemcpyDeviceToHost));
    std::cout << "[debug ] Result of scan bin_size array:" << std::endl;
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "bin[" << std::setw(3) << i << "," << std::setw(3) << j << "] = " << std::setw(2)
                      << h_binstartpts[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_binstartpts);
    std::cout << "[debug ] ---------------------------------------------------" << std::endl;
#endif
    hipEventRecord(start);
    CalcInvertofGlobalSortIdx_2d<<<(M + 1024 - 1) / 1024, 1024>>>(M, bin_size_x, bin_size_y, numbins[0], numbins[1],
                                                                  d_binstartpts, d_sortidx, d_kx, d_ky, d_idxnupts,
                                                                  pirange, nf1, nf2);
#ifdef DEBUG
    int *h_idxnupts;
    h_idxnupts = (int *)malloc(M * sizeof(int));
    checkCudaErrors(hipMemcpy(h_idxnupts, d_idxnupts, M * sizeof(int), hipMemcpyDeviceToHost));
    for (int i = 0; i < M; i++) {
        std::cout << "[debug ] idx=" << h_idxnupts[i] << std::endl;
    }
    free(h_idxnupts);
#endif
    hipEventRecord(start);
    CalcSubProb_2d<<<(M + 1024 - 1) / 1024, 1024>>>(d_binsize, d_numsubprob, maxsubprobsize, numbins[0] * numbins[1]);
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel CalcSubProb_2d\t\t%.3g ms\n", milliseconds);
#endif
#ifdef DEBUG
    int *h_numsubprob;
    h_numsubprob = (int *)malloc(n * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_numsubprob, d_numsubprob, numbins[0] * numbins[1] * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "nsub[" << std::setw(3) << i << "," << std::setw(3) << j << "] = " << std::setw(2)
                      << h_numsubprob[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    free(h_numsubprob);
#endif
    d_ptr = thrust::device_pointer_cast(d_numsubprob);
    d_result = thrust::device_pointer_cast(d_subprobstartpts + 1);
    thrust::inclusive_scan(d_ptr, d_ptr + n, d_result);
    checkCudaErrors(hipMemset(d_subprobstartpts, 0, sizeof(int)));
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Scan Subprob array\t\t%.3g ms\n", milliseconds);
#endif

#ifdef DEBUG
    printf("[debug ] Subproblem start points\n");
    int *h_subprobstartpts;
    h_subprobstartpts = (int *)malloc((n + 1) * sizeof(int));
    checkCudaErrors(hipMemcpy(h_subprobstartpts, d_subprobstartpts, (n + 1) * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < numbins[1]; j++) {
        std::cout << "[debug ] ";
        for (int i = 0; i < numbins[0]; i++) {
            if (i != 0)
                std::cout << " ";
            std::cout << "nsub[" << std::setw(3) << i << "," << std::setw(3) << j << "] = " << std::setw(2)
                      << h_subprobstartpts[i + j * numbins[0]];
        }
        std::cout << std::endl;
    }
    printf("[debug ] Total number of subproblems = %d\n", h_subprobstartpts[n]);
    free(h_subprobstartpts);
#endif
    hipEventRecord(start);
    int totalnumsubprob;
    checkCudaErrors(hipMemcpy(&totalnumsubprob, &d_subprobstartpts[n], sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMalloc(&d_subprob_to_bin, totalnumsubprob * sizeof(int)));
    MapBintoSubProb_2d<<<(numbins[0] * numbins[1] + 1024 - 1) / 1024, 1024>>>(d_subprob_to_bin, d_subprobstartpts,
                                                                              d_numsubprob, numbins[0] * numbins[1]);
    assert(d_subprob_to_bin != NULL);
    if (d_plan->subprob_to_bin != NULL)
        hipFree(d_plan->subprob_to_bin);
    d_plan->subprob_to_bin = d_subprob_to_bin;
    assert(d_plan->subprob_to_bin != NULL);
    d_plan->totalnumsubprob = totalnumsubprob;
#ifdef DEBUG
    printf("[debug ] Map Subproblem to Bins\n");
    int *h_subprob_to_bin;
    h_subprob_to_bin = (int *)malloc((totalnumsubprob) * sizeof(int));
    checkCudaErrors(
        hipMemcpy(h_subprob_to_bin, d_subprob_to_bin, (totalnumsubprob) * sizeof(int), hipMemcpyDeviceToHost));
    for (int j = 0; j < totalnumsubprob; j++) {
        std::cout << "[debug ] ";
        std::cout << "nsub[" << j << "] = " << std::setw(2) << h_subprob_to_bin[j];
        std::cout << std::endl;
    }
    free(h_subprob_to_bin);
#endif
#ifdef SPREADTIME
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Subproblem to Bin map\t\t%.3g ms\n", milliseconds);
#endif
    return 0;
}

int CUSPREAD2D_SUBPROB(int nf1, int nf2, int M, CUFINUFFT_PLAN d_plan, int blksize) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int ns = d_plan->spopts.nspread; // psi's support in terms of number of cells
    CUFINUFFT_FLT es_c = d_plan->spopts.ES_c;
    CUFINUFFT_FLT es_beta = d_plan->spopts.ES_beta;
    int maxsubprobsize = d_plan->opts.gpu_maxsubprobsize;

    // assume that bin_size_x > ns/2;
    int bin_size_x = d_plan->opts.gpu_binsizex;
    int bin_size_y = d_plan->opts.gpu_binsizey;
    int numbins[2];
    numbins[0] = ceil((CUFINUFFT_FLT)nf1 / bin_size_x);
    numbins[1] = ceil((CUFINUFFT_FLT)nf2 / bin_size_y);
#ifdef INFO
    std::cout << "[info  ] Dividing the uniform grids to bin size[" << d_plan->opts.gpu_binsizex << "x"
              << d_plan->opts.gpu_binsizey << "]" << std::endl;
    std::cout << "[info  ] numbins = [" << numbins[0] << "x" << numbins[1] << "]" << std::endl;
#endif

    CUFINUFFT_FLT *d_kx = d_plan->kx;
    CUFINUFFT_FLT *d_ky = d_plan->ky;
    CUCPX *d_c = d_plan->c;
    CUCPX *d_fw = d_plan->fw;

    int *d_binsize = d_plan->binsize;
    int *d_binstartpts = d_plan->binstartpts;
    int *d_numsubprob = d_plan->numsubprob;
    int *d_subprobstartpts = d_plan->subprobstartpts;
    int *d_idxnupts = d_plan->idxnupts;

    int totalnumsubprob = d_plan->totalnumsubprob;
    int *d_subprob_to_bin = d_plan->subprob_to_bin;

    int pirange = d_plan->spopts.pirange;

    CUFINUFFT_FLT sigma = d_plan->opts.upsampfac;
    hipEventRecord(start);

    size_t sharedplanorysize =
        (bin_size_x + 2 * (int)ceil(ns / 2.0)) * (bin_size_y + 2 * (int)ceil(ns / 2.0)) * sizeof(CUCPX);
    if (sharedplanorysize > 49152) {
        std::cout << "error: not enough shared memory" << std::endl;
        return 1;
    }

    if (d_plan->opts.gpu_kerevalmeth) {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_Subprob_Horner<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, sigma, d_binstartpts, d_binsize,
                bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize, numbins[0],
                numbins[1], d_idxnupts, pirange);
        }
    } else {
        for (int t = 0; t < blksize; t++) {
            Spread_2d_Subprob<<<totalnumsubprob, 256, sharedplanorysize>>>(
                d_kx, d_ky, d_c + t * M, d_fw + t * nf1 * nf2, M, ns, nf1, nf2, es_c, es_beta, sigma, d_binstartpts,
                d_binsize, bin_size_x, bin_size_y, d_subprob_to_bin, d_subprobstartpts, d_numsubprob, maxsubprobsize,
                numbins[0], numbins[1], d_idxnupts, pirange);
        }
    }
#ifdef SPREADTIME
    float milliseconds = 0;
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("[time  ] \tKernel Spread_2d_Subprob (%d)\t\t%.3g ms\n", milliseconds, d_plan->opts.gpu_kerevalmeth);
#endif
    return 0;
}

} // namespace spreadinterp
} // namespace cufinufft
